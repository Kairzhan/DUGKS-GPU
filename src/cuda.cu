#include <iostream>
#include <mpi.h>

#include "config.h"

int device_id;
hipDeviceProp_t prop;

//===========================================================================
// Performs basic configurations for GPU. Single-node GPU configuration is
// assumed, so that each MPI thread will select one GPU.
//===========================================================================
void init_cuda(int myrank)
{
    Config config;
    
    const int gpusPerThread=std::stoi(config.get("gpusPerThread"));;    
    device_id=myrank%gpusPerThread;
    
    hipSetDevice(device_id);
    hipGetDeviceProperties(&prop, device_id);

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n",
           device, deviceProp.major, deviceProp.minor);
    }
    
    return;
}
